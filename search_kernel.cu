#include "hip/hip_runtime.h"
#include <windows.h>
#include <GL/glew.h>

#include "searchData.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <cuda_gl_interop.h>
#include <>

__global__ void search(const SolverSettings settingData,
	const AgentData* agentData,
	hipTextureObject_t texObj,
	const Point* ptData,
	SearchOutput* outputData) 
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int agentCount = settingData.agentCount;

	if (i < agentCount && settingData.ptCount > 0) {

		float angles[3] = { 0,0,0 };
		angles[0] = agentData[i].dir + settingData.searchOffset + settingData.searchAngle;
		angles[1] = agentData[i].dir;
		angles[2] = agentData[i].dir - settingData.searchOffset - settingData.searchAngle;

		for (int dir = 0; dir < 3; dir++) {
			outputData[i].avgr[dir] = 0;
			outputData[i].avgg[dir] = 0;
			outputData[i].avgb[dir] = 0;
		}

		for (int pt = 0; pt < settingData.ptCount; pt++) {

			float ptx = (float)ptData[pt].x;
			float pty = (float)ptData[pt].y;

			for (int dir = 0; dir < 3; dir++) {

				float ptransx =
					(ptx * cosf(angles[dir]))
					- (pty * sinf(angles[dir]))
					;
				float ptransy =
					(ptx * sinf(angles[dir]))
					+ (pty * cosf(angles[dir]))
					;

				ptransx += agentData[i].x;
				ptransy += agentData[i].y;

				ptransx = (settingData.imWidth + (int)floor(ptransx)) % settingData.imWidth;
				ptransy = (settingData.imHeight + (int)floor(ptransy)) % settingData.imHeight;
				//float imPosx = ptransx / settingData.imWidth;
				//float imPosy = ptransy / settingData.imHeight;

				/*uchar4 pixelValue;
				surf2Dread(&pixelValue, surfObj, imPosx, imPosy);*/

				uchar4 pixelValue = tex2D<uchar4>(texObj, ptransx, ptransy);

				outputData[i].avgr[dir] += pixelValue.x;
				outputData[i].avgg[dir] += pixelValue.y;
				outputData[i].avgb[dir] += pixelValue.z;
			}
		}

		for (int dir = 0; dir < 3; dir++) {
			outputData[i].avgr[dir] /= settingData.ptCount;
			outputData[i].avgg[dir] /= settingData.ptCount;
			outputData[i].avgb[dir] /= settingData.ptCount;
		}
	}
}

extern "C" void runSearch(int blockSize,
	const SolverSettings settingData,
	const AgentData* dev_agentData,
	hipGraphicsResource* cudaResource,
	Point* dev_ptData,
	SearchOutput* dev_outputData) {

	hipArray_t textureArray;
	hipGraphicsMapResources(1, &cudaResource);
	hipGraphicsSubResourceGetMappedArray(&textureArray, cudaResource, 0, 0);

	/*hipResourceDesc resDesc;
	memset(&resDesc, 0, sizeof(resDesc));
	resDesc.resType = hipResourceTypeArray;
	resDesc.res.array.array = textureArray;

	hipSurfaceObject_t surfObj = 0;
	hipCreateSurfaceObject(&surfObj, &resDesc);*/

	struct hipTextureDesc texDesc;
	memset(&texDesc, 0, sizeof(texDesc));
	texDesc.addressMode[0] = hipAddressModeWrap;
	texDesc.addressMode[1] = hipAddressModeWrap;
	texDesc.filterMode = hipFilterModePoint;
	texDesc.readMode = hipReadModeElementType;
	texDesc.normalizedCoords = 0;

	hipResourceDesc resDesc;
	memset(&resDesc, 0, sizeof(resDesc));
	resDesc.resType = hipResourceTypeArray;
	resDesc.res.array.array = textureArray;

	hipTextureObject_t texObj = 0;
	hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL);

	int agentCount = settingData.agentCount;

	int gridSize = (agentCount + blockSize - 1) / blockSize;

	search << <gridSize, blockSize >> > (
		settingData,
		dev_agentData,
		texObj,
		dev_ptData,
		dev_outputData
	);

	hipDeviceSynchronize();

	/*hipDestroySurfaceObject(surfObj);*/
	hipDestroyTextureObject(texObj);

	hipGraphicsUnmapResources(1, &cudaResource);
}